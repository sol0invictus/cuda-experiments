#include <iostream>
#include <iomanip>
#include <hipblas.h>
#include "helper.cuh"
#include <hipblaslt.h>
using namespace helper;

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        throw std::logic_error("cuBLAS API failed");
    }
}


int main()
{
    //indexing in c style
    int M, N, K;
    M = 16; //rows of weight matrix
    N = 2; //batch size
    K = 16; //column of weight matrix , rows of input matrix
    // A = M x K
    // B= K x N
    // C = M x N
    // These are standard math notations


    float alpha = 1;
    float beta = 0;

	// We define our matrices and initialize them with random variables
    my_data<int8_t> A, B;
    my_data<int8_t> C;

    A.init(M*K);
    // Below A is initiazed in a cublas-trasnformed fashin aka regular cpp form 
    // this means that memory is sequentially addresssed for both A and B
    // A is stored in row major fashion now as it is transposed
	// Use this initializer to make A identity matrix
    //A.init(K * M,true,K,M); 
    B.init(K*N);
    C.init(M*N);

    hipMalloc(&A.d_ptr_, A.size_);
    hipMalloc(&B.d_ptr_, B.size_);
    hipMalloc(&C.d_ptr_, C.size_);

    
    hipMemcpy(A.d_ptr_, A.h_ptr_, A.size_, hipMemcpyHostToDevice);
    hipMemcpy(B.d_ptr_, B.h_ptr_, B.size_, hipMemcpyHostToDevice);
    hipMemcpy(C.d_ptr_, C.h_ptr_, C.size_, hipMemcpyHostToDevice);
    
    
    
    
    std::cout << "A(T):" << std::endl;
    // A matrix was transposed to need to take that into account when printing
    printMatrix(A.h_ptr_, M, K, true);
	
    std::cout << "B:" << std::endl;
    printMatrix(B.h_ptr_, K, N);
    


    void *workspace;
    size_t workspaceSize = 1024 * 1024 * 8;
    hipMalloc(&workspace, workspaceSize);
    
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    
    hipblasLtHandle_t ltHandle;
    hipblasLtCreate(&ltHandle);

    hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32I, HIP_R_32F);
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, K, M, K);
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, K, N, K);
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_8I, M, N, M);

    hipblasLtMatmulPreferenceCreate(&preference);
    
    hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
    //hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));

    hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults);
    std::cout<<"List of algos :"<<returnedResults<<std::endl;
	// create cuda event handles
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //std::cout<<cublasLtGetVersion()<<std::endl;
    hipEventRecord(start,0);
        
    hipblasLtMatmul(ltHandle,
        operationDesc,
        &alpha,
        A.d_ptr_,
        Adesc,
        B.d_ptr_,
        Bdesc,
        &beta,
        C.d_ptr_,
        Cdesc,
        C.d_ptr_,
        Cdesc,
        &heuristicResult.algo,
        workspace,
        workspaceSize,
        0);


    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float cudaElapsedTime;
    hipEventElapsedTime(&cudaElapsedTime, start, stop);
    hipMemcpy(C.h_ptr_, C.d_ptr_, C.size_, hipMemcpyDeviceToHost);

    std::cout << "C out:" << std::endl;
    printMatrix(C.h_ptr_, M, N);
    
    std::cout << std::setw(4) << cudaElapsedTime << " ms" << std::endl;
    hipFree(A.d_ptr_);
    hipFree(B.d_ptr_);
    hipFree(C.d_ptr_);
	hipblasLtDestroy(ltHandle);
    return 0;


}
